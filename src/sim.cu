#include "hip/hip_runtime.h"
#include <utility>
#include <math.h>
#include <numbers>
#include <ctime>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"


using std::pair;
using std::numbers::sqrt2;
using std::numbers::pi;
using std::rand;





//inline constexpr double R_1  = 0.501306994212753;

double distance_squared(pair<double, double> p1, pair<double, double> p2) {
    return (p1.first - p2.first)*(p1.first - p2.first) + (p1.second - p2.second)*(p1.second - p2.second);
}

double distance(pair<double, double> p1, pair<double, double> p2) {
    return sqrt((p1.first - p2.first)*(p1.first - p2.first) + (p1.second - p2.second)*(p1.second - p2.second));
}

double radius_squared(pair<double, double> p) {
    return p.first*p.first + p.second*p.second;
}

double radius(pair<double, double> p) {
    double t1 = p.first*p.first;
    double t2 = p.second*p.second;
    return sqrt(t1 + t2);
}

// Robot 1 knows the angle of the target point on the unit circle.
// It will use this to find the point that is on average closest to the target.
pair<double, double> robot1_move(double theta) {
    double r = R_1;
    pair<double, double> p = std::make_pair(r*cos(theta), r*sin(theta));
    return p;
}

pair<double, double> robot2_move(double r) {
    double theta = (rand()/(double)RAND_MAX)*2*pi;
    double robot_r;
    if (r <= R_1/2) {
        return std::make_pair(0.0, 0.0);
    } else {
        robot_r = sqrt((2*r*R_1) - (R_1*R_1));
    }

    return std::make_pair(robot_r*cos(theta), robot_r*sin(theta));
}

pair<double, double> robot2_move_const(double r) {
    double robot_r;
    if (r <= R_1/2) {
        return std::make_pair(0.0, 0.0);
    } else {
        robot_r = sqrt((2*r*R_1) - (R_1*R_1));
    }

    return std::make_pair(robot_r, 0.0);
}

pair<double, double> convert_to_polar(pair<double, double> p) {
    double r = radius(p);
    double theta = atan(p.second/p.first);
    if (p.first == 0 && p.second == 0) {  // origin
        theta = 0;
    } else if (p.first < 0) { // Q2 & Q3
        theta += pi;
    } else if (p.first > 0 && p.second < 0) { // Q4
        theta += 2*pi;
    }
    return std::make_pair(r, theta);
}

pair<double, double> convert_to_cartesian(pair<double, double> p) {
    double x = p.first * cos(p.second);
    double y = p.first * sin(p.second);
    return std::make_pair(x, y);
}



__global__ void simChunk(double* x_min, double* x_max, double* y_min, double* y_max, long int* r1_wins, long int* r2_wins, double delta) {
    int i = threadIdx.x;
    for (double x = x_min[i]; x <= x_max[i]; x += delta) {
        for (double y = y_min[i]; y_max[i] <= 1.0; y += delta) {
            pair<double, double> target = std::make_pair(x,y);
            double r;
            double theta;
            pair<double, double> target_polar = sim::convert_to_polar(target);            
            if (target_polar.first > 1.0) continue;
            trials ++;
            pair<double, double> robot_1_p = sim::robot1_move(target_polar.second);
            pair<double, double> robot_2_p = sim::robot2_move_const(target_polar.first);
            double robot_1_distance = sim::distance_squared(robot_1_p, target);
            double robot_2_distance = sim::distance_squared(robot_2_p, target);

            if (robot_1_distance < robot_2_distance) {
                robot_1_wins++;
            } else {
                robot_2_wins++;
            }
        }   
    }
}

int main(int argc, char* argv[]) {
    int accuracy = 10;
    long int trials = 0;
    for (int i = 1; i < argc; i++) {
        if (std::strcmp(argv[i], "-a") == 0) {
            if (i + 1 == argc) {
                cout << "Accuracy not given: defaulting to 10" << endl;
            } else {
                try {
                    accuracy = std::stoi(argv[i+1]);
                    if (accuracy < 1) {
                        cout << "Invalid accuracy: defaulting to 10" << endl;
                        accuracy = 10;
                    }
                } catch (std::exception &err) {
                    cout << "Invalid accuracy: defaulting to 10" << endl;
                    accuracy = 10;
                }
            }
        }
    }
    
    double delta = pow(0.5, accuracy);

    long int robot_1_wins = 0;
    long int robot_2_wins = 0;
    for (double x = 0.0; x <= 1.0; x += delta) {
        for (double y = -1.0; y <= 1.0; y += delta) {
            pair<double, double> target = std::make_pair(x,y);
            double r;
            double theta;
            pair<double, double> target_polar = sim::convert_to_polar(target);            
            if (target_polar.first > 1.0) continue;
            trials ++;
            pair<double, double> robot_1_p = sim::robot1_move(target_polar.second);
            pair<double, double> robot_2_p = sim::robot2_move_const(target_polar.first);
            double robot_1_distance = sim::distance_squared(robot_1_p, target);
            double robot_2_distance = sim::distance_squared(robot_2_p, target);

            if (robot_1_distance < robot_2_distance) {
                robot_1_wins++;
            } else {
                robot_2_wins++;
            }
        }   
    }

    double temp = sim::R_1*sim::R_1/8;

    double robot1_winrate = robot_1_wins/(double)(trials*2) + 0.5 - temp;
    double robot2_winrate = robot_2_wins/(double)(trials*2) + temp;
    
    cout << endl << std::setprecision(10) << "Robot 1 winrate: "<< robot1_winrate << endl << "Robot 2 winrate: " << robot2_winrate << endl;

    return 0;
}
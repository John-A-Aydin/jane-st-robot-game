#include <utility>
#include <math.h>
#include <numbers>
#include <ctime>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <cstring>
#include "hip/hip_runtime.h"


using std::endl;
using std::cout;

#define THREADS 512

inline constexpr double R_1  = 0.501306994212753;

__global__ void simChunk(double* y_min, double* y_max, long int* r1_wins, long int* r2_wins, double delta) {
    int i = threadIdx.x;
    for (double x = 0.0; x <= 1.0; x += delta) {
        for (double y = y_min[i]; y <= y_max[i]; y += delta) {
            double r_target = sqrt(x*x + y*y);
            if (r_target > 1.0) continue; // Outside of game area
            if (r_target <= R_1/2) { // Robot 2 wins by default
                r2_wins[i]++;
                continue;
            }
            double theta = atan(y/x);
            double r1_distance = r_target - R_1;
            // Calculating distance between robot 2 and target
            double x_r2 = sqrt((2*r_target*R_1) - (R_1*R_1));
            double r2_distance = sqrt((x - x_r2)*(x - x_r2) + y*y);
            if (r2_distance < r1_distance) {
                r2_wins[i]++;
            } else {
                r1_wins[i]++;
            }
        }   
    }
}

int main(int argc, char* argv[]) {
    int accuracy = 10;
    long int trials = 0;
    for (int i = 1; i < argc; i++) {
        if (std::strcmp(argv[i], "-a") == 0) {
            if (i + 1 == argc) {
                cout << "Accuracy not given: defaulting to 10" << endl;
            } else {
                try {
                    accuracy = std::stoi(argv[i+1]);
                    if (accuracy < 1) {
                        cout << "Invalid accuracy: defaulting to 10" << endl;
                        accuracy = 10;
                    }
                } catch (std::exception &err) {
                    cout << "Invalid accuracy: defaulting to 10" << endl;
                    accuracy = 10;
                }
            }
        }
    }
    
    double delta = pow(0.5, accuracy);

    double y_min[THREADS] = {0};
    double y_max[THREADS] = {0};
    long int r1_wins[THREADS] = {0};
    long int r2_wins[THREADS] = {0};

    for (int i = 0; i < THREADS; i++) {
        y_min[i] = (1.0/THREADS)*(double)i;
        y_max[i] = (1.0/THREADS)*(double)(i+1) - delta;
    }
    y_max[THREADS] = 1.0;
    
    double* cuda_y_min = 0;
    double* cuda_y_max = 0;
    long int* cuda_r1_wins = 0;
    long int* cuda_r2_wins = 0;

    hipMalloc(&cuda_y_min, sizeof(y_min));
    hipMalloc(&cuda_y_max, sizeof(y_max));
    hipMalloc(&cuda_r1_wins, sizeof(r1_wins));
    hipMalloc(&cuda_r2_wins, sizeof(r2_wins));

    hipMemcpy(cuda_y_min, y_min, sizeof(y_min), hipMemcpyHostToDevice);
    hipMemcpy(cuda_y_max, y_max, sizeof(y_max), hipMemcpyHostToDevice);

    simChunk <<< 1, THREADS >>> (cuda_y_min, cuda_y_max, cuda_r1_wins, cuda_r2_wins, delta);

    hipMemcpy(r1_wins, cuda_r1_wins, sizeof(r1_wins), hipMemcpyDeviceToHost);
    hipMemcpy(r2_wins, cuda_r2_wins, sizeof(r2_wins), hipMemcpyDeviceToHost);
    long int r1_total, r2_total = 0;

    for (int i = 0; i < THREADS; i++) {
        r1_total += r1_wins[i];
        r2_total += r2_wins[i];
        trials = trials + r1_wins[i] + r2_wins[i];
    }

    double temp = R_1*R_1/8;

    double robot1_winrate = r1_total/(double)(trials*2) + 0.5 - temp;
    double robot2_winrate = r2_total/(double)(trials*2) + temp;
    //cout << r1_wins[0] << endl;
    cout << endl << std::setprecision(10) << "Robot 1 winrate: "<< robot1_winrate << endl << "Robot 2 winrate: " << robot2_winrate << endl;

    return 0;
}